#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <cstddef>
#include <execution>
#include <iostream>
#include <string_view>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform_reduce.h>

namespace monaco {

  class black_scholes_analytical {
    const float sign;     // +1 for call, -1 for put
    const float s0;       // initial spot price
    const float sigma;    // volatility
    const float k;        // strike
    const float r;        // interest rate
    const float T;        // term in years

    static float cndf(float x) {
      return std::erfc(-x / std::sqrt(float(2))) / float(2);
    }

  public:
    black_scholes_analytical(const float sign,
                             const float s0,
                             const float sigma,
                             const float k,
                             const float r,
                             const float T)
        : sign(sign), s0(s0), sigma(sigma), k(k), r(r), T(T) {
    }

    float calculate() const {
      const auto d1 = (std::log(s0 / k) + (r + sigma * sigma / float(2)) * T) / (sigma * std::sqrt(T));
      const auto d2 = d1 - sigma * std::sqrt(T);
      return sign * s0 * cndf(sign * d1) - sign * std::exp(-r * T) * k * cndf(sign * d2);
    }
  };

  class black_scholes_montecarlo final : public thrust::unary_function<std::size_t, float> {
    const float sign;     // +1 for call, -1 for put
    const float s0;       // initial spot price
    const float sigma;    // volatility
    const float k;        // strike
    const float r;        // interest rate
    const float T;        // term in years
    const std::size_t num_steps;

  public:
    black_scholes_montecarlo(const float sign,
                             const float s0,
                             const float sigma,
                             const float k,
                             const float r,
                             const float T,
                             const std::size_t num_steps)
        : sign(sign), s0(s0), sigma(sigma), k(k), r(r), T(T), num_steps(num_steps) {
    }

    __host__ __device__
    float operator()(const std::size_t thread_id) const {
      thrust::default_random_engine generator(0u);
      thrust::normal_distribution<float> gaussian(0, 1);
      generator.discard(num_steps * thread_id); // don't reuse subsequences
      auto s = s0;
      const auto dt = T / static_cast<float>(num_steps);
      const auto sqrt_dt = sqrtf(dt);
      for (auto i = 0ul; i < num_steps; i++) {
        s += r * s * dt + sigma * s * sqrt_dt * gaussian(generator);
      }
      return fmaxf(sign * (s - k), float(0));
    }
  };

  template<typename Lambda>
  void time_it(const std::string_view description, Lambda&& lambda) {
    const auto start = std::chrono::system_clock::now();
    const auto return_value = lambda();
    const auto end = std::chrono::system_clock::now();
    const auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << description << return_value << " (" << millis << "ms)" << std::endl;
  }

  template<typename ExecutionPolicy, typename UnaryFunction>
  class montecarlo {
    const ExecutionPolicy& execution_policy;
    const UnaryFunction& unary_function;
    const std::size_t num_paths;

  public:
    montecarlo(const ExecutionPolicy& execution_policy,
               const UnaryFunction& unary_function,
               const std::size_t num_paths)
        : execution_policy(execution_policy)
        , unary_function(unary_function)
        , num_paths(num_paths) {
    }

    float calculate() const {
      return thrust::transform_reduce(execution_policy,
                                      thrust::counting_iterator<std::size_t>(0ul),
                                      thrust::counting_iterator<std::size_t>(num_paths),
                                      unary_function,
                                      float(),
                                      thrust::plus<float>()) / num_paths;
    }
  };

}

int main(const int argc, const char** const argv) {
  if (argc < 3) {
    std::cerr << "usage:\n\nmontecarlo <num paths> <num steps>" << std::endl;
    return -1;
  }

  const auto num_paths_long = std::atol(argv[1]);
  const auto num_steps_long = std::atol(argv[2]);

  if (num_paths_long <= 0L || num_steps_long <= 0L) {
    std::cerr << "number of paths and steps must be greater than zero" << std::endl;
    return -1;
  }

  hipDeviceSynchronize(); // warmup device

  const auto num_paths = static_cast<std::size_t>(num_paths_long);
  const auto num_steps = static_cast<std::size_t>(num_steps_long);
  const auto s0 = float(100);
  const auto sigma = float(0.2);
  const auto k = float(110);
  const auto r = float(0.01);
  const auto t = float(0.5);

  std::array<float, 2> signs {float(-1), float(1)};

  for (const auto sign : signs) {
    std::cout << "-- sign = " << sign << std::endl;

    const monaco::black_scholes_analytical analytical(sign, s0, sigma, k, r, t);
    const monaco::black_scholes_montecarlo montecarlo(sign, s0, sigma, k, r, t, num_steps);
    const monaco::montecarlo seq(thrust::seq, montecarlo, num_paths);
    const monaco::montecarlo host(thrust::host, montecarlo, num_paths);
    const monaco::montecarlo device(thrust::device, montecarlo, num_paths);

    monaco::time_it("analytical           : ", [&analytical]() { return analytical.calculate(); });
    // monaco::time_it("monte-carlo (seq)    : ", [&seq]() { return seq.calculate(); });
    // monaco::time_it("monte-carlo (host)   : ", [&host]() { return host.calculate(); });
    monaco::time_it("monte-carlo (device) : ", [&device]()   { return device.calculate(); });
  }
}
